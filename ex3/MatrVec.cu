#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>

#define THREAD_NUM 256
#define MATRIX_SIZE 1000

int blocks_num = (MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

void generateMatrix(float *a, float *b)		//a for matrix b for vector
{
	int i;
	int size = MATRIX_SIZE * MATRIX_SIZE;
	for (i = 0; i < size; i++)
		a[i] = i;
	for (i = 0; i < MATRIX_SIZE; i++)
		b[i] = i * i;
}

__global__ static void CUDAkernal(const float *a, const float *b, float *c, int n)
{
	int i;
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * THREAD_NUM + tid;
	if (idx < n)
	{
		float t = 0;
		for (i = 0; i < n; i++)
			t += a[idx*n+i]*b[i];
		c[idx] = t;
	}
}

int main()
{
	int i;
	float *a, *b, *c;
	float *cuda_a, *cuda_b, *cuda_c;
	int n = MATRIX_SIZE;
	//alloc
	a = (float*)malloc(sizeof(float)*n*n);
	b = (float*)malloc(sizeof(float)*n);
	c = (float*)malloc(sizeof(float)*n);

	hipMalloc((void**)&cuda_a, sizeof(float)*n*n);
	hipMalloc((void**)&cuda_b, sizeof(float)*n);
	hipMalloc((void**)&cuda_c, sizeof(float)*n);

	generateMatrix(a, b);

	//Copy
	hipMemcpy(cuda_a, a, sizeof(float)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float)*n, hipMemcpyHostToDevice);

	CUDAkernal<<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_c, n);
	
	hipMemcpy(c, cuda_c, sizeof(float)*n, hipMemcpyDeviceToHost);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
}

