#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>
#define THREAD_NUM 256
#define MATRIX_SIZE 1000
int blocks_num = (MATRIX_SIZE * MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

void generateMatrix(float * a, float *b)
{
	int i;
	int SIZE = MATRIX_SIZE * MATRIX_SIZE;
	for (i = 0; i < SIZE; i ++)
	{
		a[i] = i;
		b[i] = i * i;
	}
}

__global__ static void CUDAkernal(const float *a ,const float *b, float *c, int n)
{
	//block threadID
	const int tid = threadIdx.x;
	//blockID
	const int bid = blockIdx.x;
	//threadID
	const int idx = bid * THREAD_NUM + tid;
	const int row = idx / n;
	const int column = idx % n;
	if (row < n && column < n)
	{ 
		float t = 0;
		for (i = 0; i < n; i++)
		{ 
			t += a[row * n + i] * b[i * n + column];
		}
	}
}

int main()
{
	float *a, *b, *c, *d;
	float *cuda_a, * cuda_b, * cuda_c;
	int n = MATRIX_SIZE;

	//CPU alloc
	a = (float*)malloc(sizeof(float) * n * n);
	b = (float*)malloc(sizeof(float) * n * n);
	c = (float*)malloc(sizeof(float) * n * n);
	d = (float*)malloc(sizeof(float) * n * n);

	//GPU alloc
	hipMalloc((void**)&cuda_a, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_b, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_c, sizeof(flaot) * n * n);

	generateMatrix(a, b);

	//Copy
	hipMemcpy(cuda_a, a, sizeof(float)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float)*n*n, hipMemcpyHostToDevice);

	CUDAkernal << <blocks_num, THREAD_NUM, 0>> >(cuda_a, cuda_b, cuda_c, n);

	//Copy
	hipMemcpy(c, cuda_c, sizeof(float)*n*n, hipMemcpyDeviceToHost);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
}
